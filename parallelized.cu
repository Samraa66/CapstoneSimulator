#include "hip/hip_runtime.h"
%%writefile sequential.cu

#define AND  0
#define OR   1
#define NAND 2
#define NOR  3
#define XOR  4
#define XNOR 5

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void evaluate_cuda(int* a,int* b, int* op,int* result, int N)
{
    int idx = threadIdx.x;
    if (idx < N)
    {
        if (op[idx] == AND)
            result[idx] = a[idx] && b[idx];
        else if (op[idx] == OR)
            result[idx] = a[idx] || b[idx];
        else if (op[idx] == NAND)
            result[idx] = !(a[idx] && b[idx]);
        else if (op[idx] == NOR)
            result[idx] = !(a[idx] || b[idx]);
        else if (op[idx] == XOR)
            result[idx] = (a[idx] || b[idx]) && !(a[idx] && b[idx]);
        else if (op[idx] == XNOR)
            result[idx] = (!a[idx] && !b[idx]) || (a[idx] && b[idx]);
    }
}


int main(int argc,char* argv[])
{
    if(argc !=4)
    {
        printf("Error: Invalid number of arguments.\n");
        return 1;
    }
    int N = atoi(argv[2]);
    char *inputFile = argv[1];
    char *outputFile = argv[3];

    // Allocate host memory
    int *hostA = (int*)malloc(N * sizeof(int));
    int *hostB = (int*)malloc(N * sizeof(int));
    int *hostOp = (int*)malloc(N * sizeof(int));
    int *hostResult = (int*)malloc(N * sizeof(int));

    // Read input file
    FILE *fp1 = fopen(inputFile, "r");
    if (!fp1) {
        perror("Error opening input file");
        return 1;
    }
    for (int i = 0; i < N; i++) {
        fscanf(fp1, "%d,%d,%d", &hostA[i], &hostB[i], &hostOp[i]);
    }
    fclose(fp1);

    // Allocate GPU memory
    int *gpuA, *gpuB, *gpuOp, *gpuResult;
    hipMalloc((void**)&gpuA, N * sizeof(int));
    hipMalloc((void**)&gpuB, N * sizeof(int));
    hipMalloc((void**)&gpuOp, N * sizeof(int));
    hipMalloc((void**)&gpuResult, N * sizeof(int));

    // Copy data from host to GPU
    hipMemcpy(gpuA, hostA, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpuB, hostB, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpuOp, hostOp, N * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256; // Standard CUDA block size
    int gridSize = (N + blockSize - 1) / blockSize;
    // Launch kernel with N threads (1 per operation)
    evaluate_cuda<<<gridSize, blockSize>>>(gpuA, gpuB, gpuOp, gpuResult, N);
     hipDeviceSynchronize();

    // Check for CUDA errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    // Copy result from GPU to host
    hipMemcpy(hostResult, gpuResult, N * sizeof(int), hipMemcpyDeviceToHost);
    FILE *fp2 = fopen(outputFile, "w");
    if (!fp2) {
        perror("Error opening output file");
        return 1;
    }
    for (int i = 0; i < N; i++) {
        fprintf(fp2, "%d\n", hostResult[i]);
    }
    fclose(fp2);
    free(hostA); free(hostB); free(hostOp); free(hostResult);
    hipFree(gpuA); hipFree(gpuB); hipFree(gpuOp); hipFree(gpuResult);

    return 0;
}


